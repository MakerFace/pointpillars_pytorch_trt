#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>
#include "cuda_scatter.h"

namespace NAMESPACE
{
extern "C" __global__ void Scatter(const float *features_rw, const int *indices_rw, float *output_rw, 
                                    int spatialShape0, int spatialShape1, int spatialShape2,
                                    int num_voxels, int num_features)
{
    int idx    = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = idx; i < num_voxels; i += stride)
    {
        int4 coor = reinterpret_cast<const int4*>(indices_rw)[i];
        int output_vol = spatialShape0 * spatialShape1 * spatialShape2;

        //remove init -1.
        if(coor.x < 0 || coor.y < 0 || coor.z < 0 || coor.w < 0) continue;

        float *outPerBatch = output_rw + coor.x * num_features * output_vol;
        int offset = coor.y * spatialShape1 * spatialShape2 + coor.z * spatialShape2 + coor.w;

        for(int j = 0; j < num_features; ++j)
            outPerBatch[j * output_vol + offset] = features_rw[i * num_features + j];
	}

}

void cuda_scatter(const float *features_rw, const int *indices_rw,  float *output_rw, std::vector<int> spatialShape_rw,
                int num_voxels, int num_features)
{
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
                        // maximum occupancy for a full device launch
    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, Scatter, 0, num_voxels));
    minGridSize = std::min(minGridSize, DivUp(num_voxels, blockSize));

    Scatter<<<minGridSize, blockSize>>>(features_rw, indices_rw, output_rw, spatialShape_rw[0], spatialShape_rw[1], spatialShape_rw[2], num_voxels, num_features);
    //hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    checkCudaErrors(error);

}


extern "C" __global__ void Scatter_Backward(const float *features_rw, const int *indices_rw,
                                            float *output_rw, int oX, int oY, int oZ,
                                            int num_voxels, int num_features)
{
    int idx    = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = idx; i < num_voxels; i += stride)
    {
        int bs = indices_rw[i * 4];
        int x  = indices_rw[i * 4 + 1];
        int y  = indices_rw[i * 4 + 2];
        int z  = indices_rw[i * 4 + 3];

        //remove init -1.
        if(bs<0 || x<0 || y<0 || z<0) continue;

        // out shape: (bs, c, x, y, z)
        int output_vol = oX*oY*oZ;
        const float *inPerBatch = features_rw + bs * num_features * output_vol;
        int offset = x * oY * oZ + y * oZ + z;

        #pragma unroll
        for(int j = 0; j < num_features; ++j)
            output_rw[i * num_features + j] = inPerBatch[j * output_vol + offset];
	}


}
 
void cuda_scatter_backward(const float *features_rw, const int *indices_rw,  float *output_rw,
                            std::vector<int> spatialShape_rw, int num_voxels, int num_features)
{
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
                        // maximum occupancy for a full device launch
    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, Scatter, 0, num_voxels));
    minGridSize = std::min(minGridSize, DivUp(num_voxels, blockSize));

    Scatter_Backward<<<minGridSize, blockSize>>>(features_rw, indices_rw, output_rw, spatialShape_rw[0], spatialShape_rw[1], spatialShape_rw[2], num_voxels, num_features);
    //hipDeviceSynchronize();

}
 
}//namespace