#include "hip/hip_runtime.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <algorithm>

#include "dense_nova.h"
#include "plugin.h"
#include <NvInfer.h>
#include <assert.h>
#include <iostream>
#include <stdio.h>

namespace NAMESPACE
{
extern "C" __global__ void Scatter(const float *features_rw, const int *indices_rw, const int *valid_rw, float *output_rw,
                                    int spatialShape0, int spatialShape1, int spatialShape2,
                                    int max_voxels, int batch_size, int num_features)
{
    int idx    = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for(int i = idx; i < max_voxels * batch_size; i += stride)
    {
        const int batch_id = i / max_voxels;
        const int voxel_id_per_batch = i % max_voxels;
        if(voxel_id_per_batch>=valid_rw[batch_id]) continue;

        int3 coor = reinterpret_cast<const int3*>(indices_rw)[i];
        int output_vol = spatialShape0 * spatialShape1 * spatialShape2;


        float *outPerBatch = output_rw + batch_id * num_features * output_vol;
        int offset = coor.x * spatialShape1 * spatialShape2 + coor.y * spatialShape2 + coor.z;

        for(int j = 0; j < num_features; ++j)
            outPerBatch[j * output_vol + offset] = features_rw[i * num_features + j];
	}

}

void cuda_scatter(const float *features_rw, const int *indices_rw, const int *valid_rw,  float *output_rw, std::vector<int> spatialShape_rw,
                int max_voxels, int batch_size, int num_features)
{
    int blockSize;   // The launch configurator returned block size
    int minGridSize; // The minimum grid size needed to achieve the
                        // maximum occupancy for a full device launch
    checkCudaErrors(hipOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, Scatter, 0, max_voxels * batch_size));
    minGridSize = std::min(minGridSize, DivUp(max_voxels * batch_size, blockSize));

    Scatter<<<minGridSize, blockSize>>>(features_rw, indices_rw, valid_rw, output_rw, spatialShape_rw[0], spatialShape_rw[1], spatialShape_rw[2], max_voxels, batch_size, num_features);

}
 
}//namespace